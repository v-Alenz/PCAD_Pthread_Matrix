#include "hip/hip_runtime.h"
/* PCAD - Pthread Matrix
 * Prima consegna del corso di PCAD dell'Universita' degli Studi di Genova
 * Programma per la manipolazione di matrici in parallelo 
 * creata usando la libreria pthread
 *
 * Copyright (C) 2023 Andrea Valenzano 4548315
 * Copyright (C) 2023 Lorenzo Contino 4832500
 * Copyright (C) 2023 Eugenio Pallestrini 4878184
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */ 
#include "matrix.h"
#include "doomBench.h"
#include <stdio.h>

#define matrix_create_return(value) do{return_value=(value);goto matrix_create_to_return;}while(0);
Errno CUDA_matrix_create(Matrix * const matrix, uint32_t const h, uint32_t const w){
    Errno return_value = 0;
    if (h==0 && w==0) {
        matrix->w = 0;
        matrix->h = 0;
        matrix->data = NULL; 
        matrix_create_return(return_value);
    }
    if (h==0 || w==0) {
        matrix_create_return(EMATRIXSIZEISZERO)
    }
    matrix->h = h;
    matrix->w = w;
    hipMallocManaged(&matrix->data, sizeof(float*)*h);
    if(matrix->data==NULL){
        matrix_create_return(errno);
    }
    for(uint32_t i=0; i<h; i++){
        hipMallocManaged(&matrix->data[i], sizeof(float)*w);
        if(matrix->data[i]==NULL){
            matrix_create_return(errno);
        }
    }
matrix_create_to_return:
    return return_value;
}


#define matrix_init_return_random(value) do{return_value=(value);goto matrix_init_to_return_random;}while(0);
Errno CUDA_matrix_init(Matrix * const matrix, uint32_t const h, uint32_t const w, float const value){
    Errno return_value = 0;
    return_value = CUDA_matrix_create(matrix, h, w);
    if(return_value != 0){
        matrix_init_return_random(return_value);
    }
    return_value = matrix_fill(matrix, value);
    if(return_value != 0){
        matrix_init_return_random(return_value);
    }
matrix_init_to_return_random:
    return return_value;
}


void CUDA_matrix_delete(Matrix * const matrix){
    for(uint32_t i=0; i<matrix->h; i++){
        hipFree(&matrix->data[i]);
    }
    hipFree(&matrix->data);
    matrix->data = NULL;
    matrix->h=0;
    matrix->w=0;
}


#define matrix_transpose_return(value) do{return_value=(value);goto matrix_transpose_to_return;}while(0);
Errno CUDA_matrix_transpose(Matrix * const matrix){
    Errno return_value = 0;
    Matrix old_data;
    old_data.h = matrix->h;
    old_data.w = matrix->w;
    old_data.data = matrix->data;
    if((return_value=CUDA_matrix_create(matrix, old_data.w, old_data.h))!=0){
       matrix_transpose_return(return_value); 
    }
    for (uint32_t i=0; i<matrix->h; i++){
        for(uint32_t j=0; j<matrix->w; j++){
            matrix->data[i][j] = old_data.data[j][i];
        }
    }
matrix_transpose_to_return:
    CUDA_matrix_delete(&old_data);
    return return_value;
}


__global__ void CUDA_matrix_multiply_worker(Matrix const * const first, Matrix * const second, Matrix * result){
    float result_number = 0.0f;
    for(uint32_t i=0; i<result->w; i++){
        result_number += first->data[threadIdx.x/result->w][i] * second->data[threadIdx.x%result->w][i];
    }
    result->data[threadIdx.x/result->w][threadIdx.x%result->w] = result_number;
}

#define matrix_multiply_return(value) do{return_value=(value);goto matrix_multiply_to_return;}while(0);
Errno CUDA_matrix_multiply(Matrix const * const first, Matrix * const second, Matrix * result){
    Errno return_value = 0;
    if(first->w != second->h){ 
        matrix_multiply_return(EMATRIXINCONPATIBLESIZE);
    }
    if(result->data != NULL){
        CUDA_matrix_delete(result);
    }
    return_value = CUDA_matrix_create(result, first->h, second->w);
    if(return_value != 0){
        matrix_multiply_return(return_value);
    } 
    return_value = CUDA_matrix_transpose(second);
    if(return_value != 0){
        matrix_multiply_return(return_value);
    }
    CUDA_matrix_multiply_worker<<<(result->h*result->w)/256, 256>>>(first, second, result); 
matrix_multiply_to_return:
    return return_value;
}


#define M 1000
#define N 1500
#define P 1200


Matrix A, B, C, R, Rm;

void setup( void ){
    matrix_init(&A, M, N, 1.0f);
    matrix_init(&B, N, P, 1.0f);
    matrix_init(&C, P, M, 1.0f);
    matrix_create(&R, 0, 0);
    matrix_create(&Rm, 0, 0);
}
void after( void ){
    matrix_delete(&A);
    matrix_delete(&B);
    matrix_delete(&C);
    matrix_delete(&R);
    matrix_delete(&Rm);
}

void CUDA_setup( void ){
    CUDA_matrix_init(&A, M, N, 1.0f);
    CUDA_matrix_init(&B, N, P, 1.0f);
    CUDA_matrix_init(&C, P, M, 1.0f);
    CUDA_matrix_create(&R, 0, 0);
    CUDA_matrix_create(&Rm, 0, 0);
}
void CUDA_after( void ){
    CUDA_matrix_delete(&A);
    CUDA_matrix_delete(&B);
    CUDA_matrix_delete(&C);
    CUDA_matrix_delete(&R);
    CUDA_matrix_delete(&Rm);
}

void multiply_singe_thread( void ){
    matrix_multiply(&A, &B, &Rm);
    matrix_multiply(&C, &Rm, &R);
}
void multiply_multi_pthread_2( void ){
    matrix_multiply_pthread(&A, &B, &Rm, 2);
    matrix_multiply_pthread(&C, &Rm, &R, 2);
}

void multiply_multi_pthread_4( void ){
    matrix_multiply_pthread(&A, &B, &Rm, 4);
    matrix_multiply_pthread(&C, &Rm, &R, 4);
}

void multiply_multi_pthread_8( void ){
    matrix_multiply_pthread(&A, &B, &Rm, 8);
    matrix_multiply_pthread(&C, &Rm, &R, 8);
}

void multiply_multi_pthread_16( void ){
    matrix_multiply_pthread(&A, &B, &Rm, 16);
    matrix_multiply_pthread(&C, &Rm, &R, 16);
}


void cuda_multiply( void ){
    CUDA_matrix_multiply(&A, &B, &Rm);
    CUDA_matrix_multiply(&C, &Rm, &R);
}

void print_bencmark_result( Bencmark* bencmark ){
    printf("Function Name: %s\n", bencmark->function_name);
    printf("Execuiton time: %ums\n", bencmark->milliseconds_result/1000); 
    printf("===========================================================\n");
}

int main( void )
{
    BENCMARK_INIT;
    BENCMARK_SETUP(setup);
    BENCMARK_AFTER(after);
    printf("== MATRIX MULTIPLY BENCMARK ===============================\n");
    /* BENCMARK(multiply_multi_pthread_2); */
    /* print_bencmark_result(&bencmark); */
    /* BENCMARK(multiply_multi_pthread_4); */
    /* print_bencmark_result(&bencmark); */
    /* BENCMARK(multiply_multi_pthread_8); */
    /* print_bencmark_result(&bencmark); */
    BENCMARK(multiply_multi_pthread_16);
    print_bencmark_result(&bencmark);
    printf("== MATRIX MULTIPLY CUDA BENCMARK ==========================\n");
    BENCMARK_SETUP(CUDA_setup);
    BENCMARK_AFTER(CUDA_after);
    BENCMARK(cuda_multiply);
    print_bencmark_result(&bencmark);

    return EXIT_SUCCESS;
}
